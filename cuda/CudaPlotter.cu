#include "hip/hip_runtime.h"
#include "CudaPlotter.h"
#include "CudaPlotContext.h"
#include "pos/chacha8.h"
#include "b3/blake3.h"
#include "threading/MTJob.h"
#include "util/jobs/MemJobs.h"
#include "util/StackAllocator.h"
#include "CudaParkSerializer.h"
#include "plotting/CTables.h"
#include "plotting/TableWriter.h"
#include "plotting/PlotTools.h"

// TEST/DEBUG
#if _DEBUG
    #include "algorithm/RadixSort.h"
    #include "plotdisk/jobs/IOJob.h"
    #include "io/FileStream.h"

    ThreadPool* _dbgThreadPool = nullptr;

    static void DbgPruneTableBuckets( CudaK32PlotContext& cx, const TableId rTable );
    static void DbgPruneTable( CudaK32PlotContext& cx, const TableId rTable );
#endif

static void InitContext( CudaK32PlotConfig& cfg, CudaK32PlotContext*& outContext );
static void CudaInit( CudaK32PlotContext& cx );

void GenF1Cuda( CudaK32PlotContext& cx );

static void MakePlot( CudaK32PlotContext& cx );
static void FpTable( CudaK32PlotContext& cx );
static void FpTableBucket( CudaK32PlotContext& cx, const uint32 bucket );
static void UploadBucketForTable( CudaK32PlotContext& cx, const uint64 bucket );
static void FinalizeTable7( CudaK32PlotContext& cx );
static void InlineTable( CudaK32PlotContext& cx, const uint32* devInX, hipStream_t stream );

static void AllocBuffers( CudaK32PlotContext& cx );
static void AllocateP1Buffers( CudaK32PlotContext& cx, CudaK32AllocContext& acx );

template<typename T>
static void UploadBucketToGpu( CudaK32PlotContext& context, TableId table, const uint32* hostPtr, T* devPtr, uint64 bucket, uint64 stride );
static void LoadAndSortBucket( CudaK32PlotContext& cx, const uint32 bucket );

void CudaMatchBucketizedK32( CudaK32PlotContext& cx, const uint32* devY, hipStream_t stream, hipEvent_t event );

// Defined in FxCuda.cu
void GenFx( CudaK32PlotContext& cx, const uint32* devYIn, const uint32* devMetaIn, hipStream_t stream );

static const char* USAGE = "bladebit_cuda ... cudaplot <out_dir>\n"
R"(
GPU-based (CUDA) plotter

[OPTIONS]:
 -h, --help           : Shows this help message and exits.
 -d, --device         : Select the CUDA device index. (default=0)
)";

///
/// CLI
///
//-----------------------------------------------------------
void CudaK32Plotter::ParseCLI( const GlobalPlotConfig& gCfg, CliParser& cli )
{
    CudaK32PlotConfig& cfg = _cfg;
    cfg.gCfg = &gCfg;

    while( cli.HasArgs() )
    {
        if( cli.ReadU32( cfg.deviceIndex, "-d", "--device" ) )
            continue;
        if( cli.ReadSwitch( cfg.disableDirectDownloads, "--no-direct-downloads" ) )
            continue;
        if( cli.ArgMatch( "--help", "-h" ) )
        {
            Log::Line( USAGE );
            exit( 0 );
        }
        else
            break;  // Let the caller handle it
    }

    // The rest should be output directies, parsed by the global config parser.
}

//-----------------------------------------------------------
void CudaK32Plotter::Init()
{
    if( _cx )
        return;

    InitContext( _cfg, _cx );
}

//-----------------------------------------------------------
void InitContext( CudaK32PlotConfig& cfg, CudaK32PlotContext*& outContext )
{
    auto& cx = *new CudaK32PlotContext{};
    outContext = &cx;

    cx.cfg  = cfg;
    cx.gCfg = cfg.gCfg;

    Log::Line( "[Bladebit CUDA Plotter]" );
    CudaInit( cx );

    CudaErrCheck( hipStreamCreateWithFlags( &cx.computeStream , hipStreamNonBlocking ) );
    CudaErrCheck( hipStreamCreateWithFlags( &cx.computeStreamB, hipStreamNonBlocking ) );
    CudaErrCheck( hipStreamCreateWithFlags( &cx.computeStreamC, hipStreamNonBlocking ) );
    CudaErrCheck( hipStreamCreateWithFlags( &cx.computeStreamD, hipStreamNonBlocking ) );

    hipEventCreateWithFlags( &cx.computeEventA, hipEventDisableTiming );
    hipEventCreateWithFlags( &cx.computeEventB, hipEventDisableTiming );
    hipEventCreateWithFlags( &cx.computeEventC, hipEventDisableTiming );

    for( int32 i = 0; i < BBCU_GPU_STREAM_COUNT; i++ )
    {
        cx.gpuDownloadStream[i] = new GpuQueue( GpuQueue::Downloader );
        cx.gpuUploadStream  [i] = new GpuQueue( GpuQueue::Uploader   );
    }

    cx.threadPool = new ThreadPool( SysHost::GetLogicalCPUCount() );

    #if __linux__
        cx.downloadDirect = cfg.disableDirectDownloads ? false : true;
    #else
        // #TODO: One windows, check if we have enough memory, if so, default to true.
        cx.downloadDirect = true ;//false;
    #endif

    // cx.plotWriter = new PlotWriter( !cfg.gCfg->disableOutputDirectIO );
    // if( cx.gCfg->benchmarkMode )
    //     cx.plotWriter->EnableDummyMode();

    cx.plotFence  = new Fence();

    cx.phase2     = new CudaK32Phase2{};
    cx.phase3     = new CudaK32Phase3{};

    // #TODO: Support non-warm starting
    Log::Line( "Allocating buffers (this may take a few seconds)..." );
    AllocBuffers( cx );
    InitFSEBitMask( cx );
}

//-----------------------------------------------------------
void CudaInit( CudaK32PlotContext& cx )
{
    ASSERT( cx.cudaDevice == -1 );

    // CUDA init
    int deviceCount = 0;
    CudaFatalCheckMsg( hipGetDeviceCount( &deviceCount ), "Failed to fetch CUDA devices." );
    FatalIf( deviceCount < 1, "No CUDA-capable devices found." );
    FatalIf( cx.cfg.deviceIndex >= deviceCount, "CUDA device %u is out of range out of %d CUDA devices", 
            cx.cfg.deviceIndex, deviceCount );
    
    CudaFatalCheckMsg( hipSetDevice( (int)cx.cfg.deviceIndex ), "Failed to set cuda device at index %u", cx.cfg.deviceIndex );
    cx.cudaDevice = (int32)cx.cfg.deviceIndex;

    hipDeviceProp_t* cudaDevProps = new hipDeviceProp_t{};
    CudaErrCheck( hipGetDeviceProperties( cudaDevProps, cx.cudaDevice ) );
    cx.cudaDevProps = cudaDevProps;

    Log::Line( "Selected cuda device %u : %s", cx.cudaDevice, cudaDevProps->name );

    // Get info & limites
    size_t stack = 0, memFree = 0, memTotal = 0;
    hipMemGetInfo( &memFree, &memTotal );
    hipDeviceGetLimit( &stack, hipLimitStackSize );

    Log::Line( " CUDA Compute Capability   : %u.%u", cudaDevProps->major, cudaDevProps->minor );
    Log::Line( " SM count                  : %d", cudaDevProps->multiProcessorCount );
    Log::Line( " Max blocks per SM         : %d", cudaDevProps->maxBlocksPerMultiProcessor );
    Log::Line( " Max threads per SM        : %d", cudaDevProps->maxThreadsPerMultiProcessor );
    Log::Line( " Async Engine Count        : %d", cudaDevProps->asyncEngineCount );
    Log::Line( " L2 cache size             : %.2lf MB", (double)cudaDevProps->l2CacheSize BtoMB );
    Log::Line( " L2 persist cache max size : %.2lf MB", (double)cudaDevProps->persistingL2CacheMaxSize BtoMB );
    Log::Line( " Stack Size                : %.2lf KB", (double)stack   BtoKB );
    Log::Line( " Memory:" );
    Log::Line( "  Total                    : %.2lf GB", (double)memTotal BtoGB );
    Log::Line( "  Free                     : %.2lf GB", (double)memFree  BtoGB );
    Log::Line( "" );

    // Ensure we have the correct capabilities    
    //int supportsCoopLaunch = 0;
    //hipDeviceGetAttribute( &supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, cx.cudaDevice );
    //FatalIf( supportsCoopLaunch != 1, "This CUDA device does not support cooperative kernel launches." );
}


///
/// Plotting entry point
///
//-----------------------------------------------------------
void CudaK32Plotter::Run( const PlotRequest& req )
{
    SysHost::InstallCrashHandler();

    // Initialize if needed
    if( _cx == nullptr )
        Init();

    auto&       cx  = *_cx;
    const auto& cfg = _cfg;

    // Only start profiling from here (don't profile allocations)
    CudaErrCheck( hipProfilerStart() );

    ASSERT( cx.plotWriter == nullptr );
    cx.plotWriter = new PlotWriter( !cfg.gCfg->disableOutputDirectIO );
    if( cx.gCfg->benchmarkMode )
        cx.plotWriter->EnableDummyMode();

    FatalIf( !cx.plotWriter->BeginPlot( cfg.gCfg->compressionLevel > 0 ? PlotVersion::v2_0 : PlotVersion::v1_0, 
            req.outDir, req.plotFileName, req.plotId, req.memo, req.memoSize, cfg.gCfg->compressionLevel ), 
        "Failed to open plot file with error: %d", cx.plotWriter->GetError() );

    cx.plotRequest = req;
    MakePlot( cx );

    cx.plotWriter->EndPlot( true );

    // #TODO: Ensure the last plot ended here for now
    {
        const auto pltoCompleteTimer = TimerBegin();
        cx.plotWriter->WaitForPlotToComplete();
        const double plotIOTime = TimerEnd( pltoCompleteTimer );
        Log::Line( "Completed writing plot in %.2lf seconds", plotIOTime );

        cx.plotWriter->DumpTables();
    }
    Log::Line( "" );

    delete cx.plotWriter;
    cx.plotWriter = nullptr;
}

//-----------------------------------------------------------
void MakePlot( CudaK32PlotContext& cx )
{
    memset( cx.bucketCounts    , 0, sizeof( cx.bucketCounts ) );
    memset( cx.bucketSlices    , 0, sizeof( cx.bucketSlices ) );
    memset( cx.tableEntryCounts, 0, sizeof( cx.tableEntryCounts ) );

    cx.table = TableId::Table1;
    const auto plotTimer = TimerBegin();
    const auto p1Timer   = plotTimer;

    #if BBCU_DBG_SKIP_PHASE_1
        DbgLoadContextAndPairs( cx );
    #else
    // F1
    Log::Line( "Generating F1" );
    const auto timer = TimerBegin();
    GenF1Cuda( cx );
    const auto elapsed = TimerEnd( timer );
    Log::Line( "Finished F1 in %.2lf seconds.", elapsed );

    // Time for FP   
    for( TableId table = TableId::Table2; table <= TableId::Table7; table++ )
    {
        cx.table  = table;
        cx.bucket = 0;
        FpTable( cx );
    }
    const auto p1Elapsed = TimerEnd( p1Timer );
    Log::Line( "Completed Phase 1 in %.2lf seconds", p1Elapsed );
    #endif

    // Prune
    #if !BBCU_DBG_SKIP_PHASE_2
    const auto p2Timer = TimerBegin();
    CudaK32PlotPhase2( cx );
    const auto p2Elapsed = TimerEnd( p2Timer );
    Log::Line( "Completed Phase 2 in %.2lf seconds", p2Elapsed );
    #endif

    // Compress & write plot tables
    const auto p3Timer = TimerBegin();
    CudaK32PlotPhase3( cx );
    const auto p3Elapsed = TimerEnd( p3Timer );
    Log::Line( "Completed Phase 3 in %.2lf seconds", p3Elapsed );

    auto plotElapsed = TimerEnd( plotTimer );
    Log::Line( "Completed Plot 1 in %.2lf seconds ( %.2lf minutes )", plotElapsed, plotElapsed / 60.0 );
    Log::Line( "" );
}

//-----------------------------------------------------------
void FpTable( CudaK32PlotContext& cx )
{
    memset( &cx.timings, 0, sizeof( cx.timings ) );
    const TableId inTable = cx.table - 1;

    cx.prevTablePairOffset = 0;

    // Clear slice counts
    CudaErrCheck( hipMemsetAsync( cx.devSliceCounts, 0, sizeof( uint32 ) * BBCU_BUCKET_COUNT * BBCU_BUCKET_COUNT, cx.computeStream ) );

    // Load initial buckets
    UploadBucketForTable( cx, 0 );

    const auto timer = TimerBegin();
    for( uint32 bucket = 0; bucket < BBCU_BUCKET_COUNT; bucket++ )
    {
        FpTableBucket( cx, bucket );
    }

    CudaErrCheck( hipStreamSynchronize( cx.computeStream ) );

    // Copy bucket slices to host
    hipMemcpyAsync( cx.hostBucketSlices, cx.devSliceCounts, sizeof( uint32 ) * BBCU_BUCKET_COUNT * BBCU_BUCKET_COUNT, 
                        hipMemcpyDeviceToHost, cx.gpuDownloadStream[0]->GetStream() );
    CudaErrCheck( hipStreamSynchronize( cx.gpuDownloadStream[0]->GetStream() ) );

    // #TODO: Don't do this copy and instead just use the hostBucketSlices one
    const uint32 outIdx = CudaK32PlotGetOutputIndex( cx );
    memcpy( &cx.bucketSlices[outIdx], cx.hostBucketSlices, sizeof( uint32 ) * BBCU_BUCKET_COUNT * BBCU_BUCKET_COUNT );

    // #TODO: Do this on the GPU and simply copy it over
    for( uint32 i = 0; i < BBCU_BUCKET_COUNT; i++ )
        for( uint32 j = 0; j < BBCU_BUCKET_COUNT; j++ )
            cx.bucketCounts[(int)cx.table][i] += cx.bucketSlices[outIdx][j][i];

    cx.tableEntryCounts[(int)cx.table] = 0;
    for( uint32 i = 0; i < BBCU_BUCKET_COUNT; i++ )
        cx.tableEntryCounts[(int)cx.table] += cx.bucketCounts[(int)cx.table][i];

    // Cap entry counts to 2^k
    if( cx.tableEntryCounts[(int)cx.table] > BBCU_TABLE_ENTRY_COUNT )
    {
        const uint32 overflow = (uint32)( cx.tableEntryCounts[(int)cx.table] - BBCU_TABLE_ENTRY_COUNT );
        cx.tableEntryCounts[(int)cx.table] = BBCU_TABLE_ENTRY_COUNT;
        cx.bucketCounts[(int)cx.table][BBCU_BUCKET_COUNT-1] -= overflow;
    }

    cx.yOut.WaitForCompletion();
    cx.yOut.Reset();
    
    cx.xPairsOut.WaitForCompletion();
    cx.xPairsOut.Reset();

    cx.xPairsIn.Reset();

    cx.pairsLOut.WaitForCompletion();
    cx.pairsLOut.Reset();
    cx.pairsROut.WaitForCompletion();
    cx.pairsROut.Reset();

    // #NOTE: Must do this to ensure the buffers are
    //        free for the next go, which use the same underlying buffers
    //        but a different downloader object.
    cx.sortedXPairsOut.WaitForCompletion();
    cx.sortedXPairsOut.Reset();

    cx.sortedPairsLOut.WaitForCompletion();//cx.sortedPairsLOut.WaitForCopyCompletion();
    cx.sortedPairsLOut.Reset();
    cx.sortedPairsROut.WaitForCompletion();//cx.sortedPairsROut.WaitForCopyCompletion();
    cx.sortedPairsROut.Reset();

    
    if( cx.table < TableId::Table7 )
    {
        cx.metaOut.WaitForCompletion(); cx.metaOut.Reset();
    }

    cx.yIn     .Reset();
    cx.pairsLIn.Reset();
    cx.pairsRIn.Reset();
    cx.metaIn  .Reset();

    const auto elapsed = TimerEnd( timer );
    Log::Line( "Table %u completed in %.2lf seconds with %llu entries.", 
               (uint32)cx.table+1, elapsed, cx.tableEntryCounts[(int)cx.table] );

    #if DBG_BBCU_P1_WRITE_PAIRS
        // Write them sorted, so have to wait until table 3 completes
        if( cx.table > TableId::Table2 )
            DbgWritePairs( cx, cx.table - 1 );
    #endif
    
    if( cx.table == TableId::Table7 )
    {
       FinalizeTable7( cx );

       #if DBG_BBCU_P1_WRITE_PAIRS
           DbgWritePairs( cx, TableId::Table7 );
       #endif

        #if DBG_BBCU_P1_WRITE_CONTEXT
           DbgWriteContext( cx );
       #endif
    }
}

//-----------------------------------------------------------
void FpTableBucket( CudaK32PlotContext& cx, const uint32 bucket )
{
    cx.bucket = bucket;

    // Load next bucket in the background
    if( bucket + 1 < BBCU_BUCKET_COUNT )
        UploadBucketForTable( cx, bucket + 1 );

    const TableId inTable    = cx.table - 1;
    const uint32  entryCount = cx.bucketCounts[(int)inTable][bucket];

    // #NOTE: Ensure these match the ones in UploadBucketForTable()
    hipStream_t mainStream  = cx.computeStream;
    hipStream_t metaStream  = cx.computeStream;//B;
    hipStream_t pairsStream = cx.computeStream;//C;

    uint32* sortKeyIn   = (uint32*)cx.devMatches;
    uint32* sortKeyOut  = cx.devSortKey;
    if( cx.table > TableId::Table2 )
    {
        // Generate a sorting key
        CudaK32PlotGenSortKey( entryCount, sortKeyIn, mainStream );
    }
 
    uint32* devYUnsorted    = (uint32*)cx.yIn.GetUploadedDeviceBuffer( mainStream );
    uint32* devMetaUnsorted = nullptr;

    uint32* devYSorted      = cx.devYWork;
    uint32* devMetaSorted   = cx.devMetaWork;

    if( cx.table == TableId::Table2 )
    {
        devMetaUnsorted = (uint32*)cx.metaIn.GetUploadedDeviceBuffer( mainStream );
        sortKeyIn       = devMetaUnsorted;
        sortKeyOut      = devMetaSorted;
    }

    // Sort y w/ key
    CudaErrCheck( hipcub::DeviceRadixSort::SortPairs<uint32, uint32>( 
        cx.devSortTmp, cx.devSortTmpAllocSize, 
        devYUnsorted,  devYSorted, 
        sortKeyIn,     sortKeyOut, 
        entryCount, 0, 32, mainStream ) );

    CudaErrCheck( hipEventRecord( cx.computeEventC, mainStream ) );
    CudaErrCheck( hipEventRecord( cx.computeEventA, mainStream ) );

    cx.yIn.ReleaseDeviceBuffer( mainStream );
    if( cx.table == TableId::Table2 )
        cx.metaIn.ReleaseDeviceBuffer( mainStream );

    // Sort and download prev table's pairs
    const bool isLTableInlineable = cx.table == TableId::Table2 || (uint32)cx.table <= cx.gCfg->numDroppedTables+1;
    
    if( !isLTableInlineable )
    {
        CudaErrCheck( hipStreamWaitEvent( pairsStream, cx.computeEventC ) );   // Ensure sort key is ready

        const bool isLTableInlinedPairs = (uint32)cx.table == cx.gCfg->numDroppedTables + 2;

        if( isLTableInlinedPairs )
        {
            // Table 2's pairs are inlined x's. Treat as Pairs
            Pair* pairsIn     = (Pair*)cx.xPairsIn.GetUploadedDeviceBuffer( pairsStream );
            Pair* sortedPairs = (Pair*)cx.sortedXPairsOut.LockDeviceBuffer( pairsStream );

            CudaK32PlotSortByKey( entryCount, sortKeyOut, pairsIn, sortedPairs, pairsStream );
            cx.xPairsIn.ReleaseDeviceBuffer( pairsStream );

            Pair* hostPairs = ((Pair*)cx.hostBackPointers[(int)cx.table-1].left) + cx.prevTablePairOffset;

            // Write sorted pairs back to host
            cx.sortedXPairsOut.DownloadT( hostPairs, entryCount, pairsStream, cx.downloadDirect );
        }
        else
        {
            uint32* hostPairsL, *hostPairsLFinal;
            uint16* hostPairsR, *hostPairsRFinal;

            // Wait for pairs to complete loading and sort on Y (or do this before match? Giving us time to write to disk while matching?)
            uint32* pairsLIn     = (uint32*)cx.pairsLIn       .GetUploadedDeviceBuffer( pairsStream );
            uint32* sortedPairsL = (uint32*)cx.sortedPairsLOut.LockDeviceBuffer( pairsStream );
            CudaK32PlotSortByKey( entryCount, sortKeyOut, pairsLIn, sortedPairsL, pairsStream );
            cx.pairsLIn.ReleaseDeviceBuffer( pairsStream );
            hostPairsL      = cx.hostTableSortedL + cx.prevTablePairOffset;
            hostPairsLFinal = cx.hostBackPointers[(int)cx.table-1].left  + cx.prevTablePairOffset;

            cx.sortedPairsLOut.DownloadT( hostPairsLFinal, entryCount, pairsStream, cx.downloadDirect );
            // cx.sortedPairsLOut.DownloadAndCopyT( hostPairsL, hostPairsLFinal, entryCount, pairsStream );
            
            // if( !isOutputCompressed )
            {
                uint16* pairsRIn     = (uint16*)cx.pairsRIn       .GetUploadedDeviceBuffer( pairsStream );
                uint16* sortedPairsR = (uint16*)cx.sortedPairsROut.LockDeviceBuffer( pairsStream );
                CudaK32PlotSortByKey( entryCount, sortKeyOut, pairsRIn, sortedPairsR, pairsStream );
                cx.pairsRIn.ReleaseDeviceBuffer( pairsStream );
                hostPairsR      = cx.hostTableSortedR + cx.prevTablePairOffset; 
                hostPairsRFinal = cx.hostBackPointers[(int)cx.table-1].right + cx.prevTablePairOffset;
                
                cx.sortedPairsROut.DownloadT( hostPairsRFinal, entryCount, pairsStream, cx.downloadDirect );
                // cx.sortedPairsROut.DownloadAndCopyT( hostPairsR, hostPairsRFinal, entryCount, pairsStream );
            }
        }
    }

    // Match pairs
    CudaMatchBucketizedK32( cx, devYSorted, mainStream, nullptr );

    // Inline input x's or compressed x's
    if( isLTableInlineable )
    {
        uint32* inlineInput = devMetaSorted;

        if( cx.table > TableId::Table2 )
        {
            uint32* pairsLIn = (uint32*)cx.pairsLIn.GetUploadedDeviceBuffer( pairsStream );
            inlineInput = cx.devXInlineInput;

            CudaK32PlotSortByKey( entryCount, sortKeyOut, pairsLIn, inlineInput, pairsStream );
            cx.pairsLIn.ReleaseDeviceBuffer( pairsStream );
        }

        // Inline x values into our new pairs (merge L table into R table)
        InlineTable( cx, inlineInput, mainStream );
    }

    // Upload and sort metadata
    if( cx.table > TableId::Table2 )
    {
        const uint32 metaMultiplier = GetTableMetaMultiplier( cx.table - 1 );

        // Wait for meta to complete loading, and sort on Y
        devMetaUnsorted = (uint32*)cx.metaIn.GetUploadedDeviceBuffer( metaStream );

        // Ensure the sort key is ready
        CudaErrCheck( hipStreamWaitEvent( metaStream, cx.computeEventA ) );

        switch( metaMultiplier )
        {
            case 2: CudaK32PlotSortByKey( entryCount, sortKeyOut, (K32Meta2*)devMetaUnsorted, (K32Meta2*)devMetaSorted, metaStream ); break;
            case 3: CudaK32PlotSortByKey( entryCount, sortKeyOut, (K32Meta3*)devMetaUnsorted, (K32Meta3*)devMetaSorted, metaStream ); break;
            case 4: CudaK32PlotSortByKey( entryCount, sortKeyOut, (K32Meta4*)devMetaUnsorted, (K32Meta4*)devMetaSorted, metaStream ); break;
            default: ASSERT( 0 ); break;
        }
        cx.metaIn.ReleaseDeviceBuffer( metaStream );
        CudaErrCheck( hipEventRecord( cx.computeEventB, metaStream ) );
    }

    // Ensure metadata is sorted
    CudaErrCheck( hipStreamWaitEvent( mainStream, cx.computeEventB ) );

    // Compute Fx
    GenFx( cx, devYSorted, devMetaSorted, mainStream );

    CudaK32PlotDownloadBucket( cx );

    cx.prevTablePairOffset += entryCount;
}

//-----------------------------------------------------------
void FinalizeTable7( CudaK32PlotContext& cx )
{
    Log::Line( "Finalizing Table 7" );
    
    const auto timer = TimerBegin();

    cx.table               = TableId::Table7+1;   // Set a false table
    cx.prevTablePairOffset = 0;

    // Upload initial bucket
    UploadBucketForTable( cx, 0 );


    // Prepare C1 & 2 tables
    const uint32 c1Interval       = kCheckpoint1Interval;
    const uint32 c2Interval       = kCheckpoint1Interval * kCheckpoint2Interval;

    const uint64 tableLength      = cx.tableEntryCounts[(int)TableId::Table7];
    const uint32 c1TotalEntries   = (uint32)CDiv( tableLength, (int)c1Interval ) + 1; // +1 because chiapos adds an extra '0' entry at the end
    const uint32 c2TotalEntries   = (uint32)CDiv( tableLength, (int)c2Interval ) + 1; // +1 because we add a short-circuit entry to prevent C2 lookup overflows

    const size_t c1TableSizeBytes = c1TotalEntries * sizeof( uint32 );
    const size_t c2TableSizeBytes = c2TotalEntries * sizeof( uint32 );


    // Prepare host allocations
    constexpr size_t c3ParkSize = CalculateC3Size();

    const uint64 totalParkSize = CDivT( tableLength, (uint64)kCheckpoint1Interval ) * c3ParkSize;

    StackAllocator hostAlloc( cx.hostMeta, BBCU_TABLE_ALLOC_ENTRY_COUNT * sizeof( uint32 ) * 4 );
    uint32* hostC1Buffer        = hostAlloc.CAlloc<uint32>( c1TotalEntries );
    uint32* hostC2Buffer        = hostAlloc.CAlloc<uint32>( c2TotalEntries );
    uint32* hostLastParkEntries = hostAlloc.CAlloc<uint32>( kCheckpoint1Interval );
    byte*   hostLastParkBuffer  = (byte*)hostAlloc.CAlloc<uint32>( kCheckpoint1Interval );
    byte*   hostCompressedParks = hostAlloc.AllocT<byte>( totalParkSize );
    
    byte*   hostParkWriter      = hostCompressedParks;
    uint32* hostC1Writer        = hostC1Buffer;

    // Prepare device allocations
    constexpr size_t devAllocatorSize = BBCU_BUCKET_ALLOC_ENTRY_COUNT * BBCU_HOST_META_MULTIPLIER * sizeof( uint32 );
    StackAllocator devAlloc( cx.devMetaWork, devAllocatorSize );

    constexpr uint32 maxParksPerBucket = CuCDiv( BBCU_BUCKET_ENTRY_COUNT, kCheckpoint1Interval ) + 1;
    static_assert( maxParksPerBucket * c3ParkSize < devAllocatorSize );

    uint32* devC1Buffer = devAlloc.CAlloc<uint32>( c1TotalEntries );
    uint32* devC1Writer = devC1Buffer;

    const size_t parkBufferSize = kCheckpoint1Interval * sizeof( uint32 );

    GpuDownloadBuffer& parkDownloader = cx.metaOut;

    hipStream_t mainStream     = cx.computeStream;
    hipStream_t metaStream     = cx.computeStream;//B;
    hipStream_t pairsStream    = cx.computeStream;//C;
    hipStream_t downloadStream = cx.gpuDownloadStream[0]->GetStream();

    // Load CTable
    FSE_CTable* devCTable = devAlloc.AllocT<FSE_CTable>( sizeof( CTable_C3 ), sizeof( uint64 ) );
    CudaErrCheck( hipMemcpyAsync( devCTable, CTable_C3, sizeof( CTable_C3 ), hipMemcpyHostToDevice, cx.computeStream ) );


    // Prepare plot tables
    cx.plotWriter->ReserveTableSize( PlotTable::C1, c1TableSizeBytes );
    cx.plotWriter->ReserveTableSize( PlotTable::C2, c2TableSizeBytes );
    cx.plotWriter->BeginTable( PlotTable::C3 );

    // Save a buffer with space before the start of it for us to copy retained entries for the next park.
    uint32  retainedC3EntryCount = 0;
    uint32* devYSorted           = cx.devYWork + kCheckpoint1Interval;

    
    uint32* sortKeyIn  = (uint32*)cx.devMatches;
    uint32* sortKeyOut = cx.devSortKey;

    // Compress parks
    for( uint32 bucket = 0; bucket < BBCU_BUCKET_COUNT; bucket++ )
    {
        cx.bucket = bucket;

        // Upload next bucket
        if( bucket + 1 < BBCU_BUCKET_COUNT )
            UploadBucketForTable( cx, bucket+1 );

        const uint32 entryCount = cx.bucketCounts[(int)TableId::Table7][bucket];
        ASSERT( entryCount > kCheckpoint1Interval );


        // Generate a sorting key
        CudaK32PlotGenSortKey( entryCount, sortKeyIn, mainStream );

        // Sort y w/ key
        uint32* devYUnsorted = (uint32*)cx.yIn.GetUploadedDeviceBuffer( mainStream );

        CudaErrCheck( hipcub::DeviceRadixSort::SortPairs<uint32, uint32>( 
            cx.devSortTmp, cx.devSortTmpAllocSize, 
            devYUnsorted, devYSorted,
            sortKeyIn, sortKeyOut, 
            entryCount, 0, 32, mainStream ) );

        CudaErrCheck( hipEventRecord( cx.computeEventA, mainStream ) );
        cx.yIn.ReleaseDeviceBuffer( mainStream ); devYUnsorted = nullptr;

        // Sort pairs
        {
            CudaErrCheck( hipStreamWaitEvent( pairsStream, cx.computeEventA ) );   // Wait for the sort key to be ready

            uint32* sortedPairsL = (uint32*)cx.sortedPairsLOut.LockDeviceBuffer( pairsStream );
            uint32* pairsLIn     = (uint32*)cx.pairsLIn.GetUploadedDeviceBuffer( pairsStream );
            CudaK32PlotSortByKey( entryCount, sortKeyOut, pairsLIn, sortedPairsL, pairsStream );
            cx.pairsLIn.ReleaseDeviceBuffer( pairsStream );

            uint16* sortedPairsR = (uint16*)cx.sortedPairsROut.LockDeviceBuffer( pairsStream );
            uint16* pairsRIn     = (uint16*)cx.pairsRIn.GetUploadedDeviceBuffer( pairsStream );
            CudaK32PlotSortByKey( entryCount, sortKeyOut, pairsRIn, sortedPairsR, pairsStream );
            cx.pairsRIn.ReleaseDeviceBuffer( pairsStream );


            // Download sorted pairs back to host
            // uint32* hostPairsL      = cx.hostTableSortedL + cx.prevTablePairOffset;
            // uint16* hostPairsR      = cx.hostTableSortedR + cx.prevTablePairOffset;
            uint32* hostPairsLFinal = cx.hostBackPointers[(int)TableId::Table7].left  + cx.prevTablePairOffset;
            uint16* hostPairsRFinal = cx.hostBackPointers[(int)TableId::Table7].right + cx.prevTablePairOffset;

            // cx.sortedPairsLOut.DownloadAndCopyT( hostPairsL, hostPairsLFinal, entryCount, pairsStream );
            // cx.sortedPairsROut.DownloadAndCopyT( hostPairsR, hostPairsRFinal, entryCount, pairsStream );
            cx.sortedPairsLOut.DownloadT( hostPairsLFinal, entryCount, pairsStream, true );
            cx.sortedPairsROut.DownloadT( hostPairsRFinal, entryCount, pairsStream, true );

            cx.prevTablePairOffset += entryCount;
        }


        // If we previously had entries retained, adjust our buffer and counts accordingly
        uint32* devF7Entries = devYSorted - retainedC3EntryCount;
        uint32  f7EntryCount = entryCount + retainedC3EntryCount;

        const uint32 parkCount = f7EntryCount / kCheckpoint1Interval;

        // Copy C1 entries
        CudaErrCheck( hipMemcpy2DAsync( devC1Writer, sizeof( uint32 ), devF7Entries, sizeof( uint32 ) * c1Interval,
                                         sizeof( uint32 ), parkCount, hipMemcpyDeviceToDevice, mainStream ) );
        devC1Writer += parkCount;

        // Compress C tables
        // This action mutates the f7 buffer in-place, so ensure the C1 copies happen before this call
        byte* devParkBuffer = (byte*)parkDownloader.LockDeviceBuffer( mainStream );
        CompressC3ParksInGPU( parkCount, devF7Entries, devParkBuffer, c3ParkSize, devCTable, mainStream );

        // Retain any new f7 entries for the next bucket, if ndeeded
        retainedC3EntryCount = f7EntryCount - (parkCount * kCheckpoint1Interval);
        if( retainedC3EntryCount > 0 )
        {
            // Last bucket?
            const bool isLastBucket = bucket + 1 == BBCU_BUCKET_COUNT;

            const uint32  compressedEntryCount = parkCount * kCheckpoint1Interval;
            const uint32* copySource           = devF7Entries + compressedEntryCount;
            const size_t  copySize             = sizeof( uint32 ) * retainedC3EntryCount;

            if( !isLastBucket )
            {
                // Not the last bucket, so retain entries for the next GPU compression bucket
                CudaErrCheck( hipMemcpyAsync( devYSorted - retainedC3EntryCount, copySource, copySize, 
                                                hipMemcpyDeviceToDevice, mainStream ) );
            }
            else
            {
                // No more buckets so we have to compress this last park on the CPU
                CudaErrCheck( hipMemcpyAsync( hostLastParkEntries, copySource, copySize, 
                                                hipMemcpyDeviceToHost, downloadStream ) );
            }
        }

        // Download compressed parks to host
        const size_t parkDownloadSize = c3ParkSize * parkCount;
        parkDownloader.DownloadWithCallback( hostParkWriter, parkDownloadSize, 
            []( void* parksBuffer, size_t size, void* userData ) {

                auto& cx = *reinterpret_cast<CudaK32PlotContext*>( userData );
                cx.plotWriter->WriteTableData( parksBuffer, size );
            }, &cx, mainStream );
        hostParkWriter += parkDownloadSize;
    }

    // Download c1 entries
    const size_t devC1EntryCount = (size_t)(uintptr_t)(devC1Writer - devC1Buffer);
    CudaErrCheck( hipMemcpyAsync( hostC1Buffer, devC1Buffer, sizeof( uint32 ) * devC1EntryCount, hipMemcpyDeviceToHost, downloadStream ) );
    hostC1Writer += devC1EntryCount;

    // Wait for parks to finish downloading
    parkDownloader.WaitForCompletion();
    parkDownloader.Reset();

    // Was there a left-over park?
    if( retainedC3EntryCount > 0 )
    {
        // Copy c1 entry
        *hostC1Writer++ = hostLastParkEntries[0];
        ASSERT( hostC1Writer - hostC1Buffer == c1TotalEntries - 1 );

        // Serialize and trailing park and submit it to the plot
        if( retainedC3EntryCount > 1 )
        {
            TableWriter::WriteC3Park( retainedC3EntryCount - 1, hostLastParkEntries, hostLastParkBuffer );
            cx.plotWriter->WriteTableData( hostLastParkBuffer, c3ParkSize );
        }
    }

    // Write final empty C entries
    hostC1Buffer[c1TotalEntries-1] = 0;
    hostC2Buffer[c2TotalEntries-1] = 0;

    // Byte-swap C1 
    for( uint32 i = 0; i < c1TotalEntries-1; i++ )
        hostC1Buffer[i] = Swap32( hostC1Buffer[i] );

    // Calculate C2 entries
    for( uint32 i = 0; i < c2TotalEntries-1; i++ )
    {
        ASSERT( i * kCheckpoint2Interval < c1TotalEntries - 1 );
        hostC2Buffer[i] = hostC1Buffer[i * kCheckpoint2Interval];
    }

    // End C3 table & write C1 & C2 tables
    cx.plotWriter->EndTable();
    cx.plotWriter->WriteReservedTable( PlotTable::C1, hostC1Buffer );
    cx.plotWriter->WriteReservedTable( PlotTable::C2, hostC2Buffer );
    cx.plotWriter->SignalFence( *cx.plotFence );    // Signal the fence for the start of Phase 3 when we have to use our tmp2 host buffer again


    // Cleanup
    // cx.sortedPairsLOut.WaitForCopyCompletion();
    // cx.sortedPairsROut.WaitForCopyCompletion();
    cx.sortedPairsLOut.WaitForCompletion();
    cx.sortedPairsROut.WaitForCompletion();
    cx.sortedPairsLOut.Reset();
    cx.sortedPairsROut.Reset();

    cx.prevTablePairOffset = 0;

    auto elapsed = TimerEnd( timer );
    Log::Line( "Finalized Table 7 in %.2lf seconds.", elapsed );
}

//-----------------------------------------------------------
__global__ void CudaInlineTable( const uint32* entryCount, const uint32* inX, const Pair* matches, Pair* inlinedPairs, uint32 entryBits = 0 )
{
    const uint32 gid = blockIdx.x * blockDim.x + threadIdx.x;

    if( gid >= *entryCount )
        return;

    const Pair pair = matches[gid];

    Pair inlined;
    inlined.left  = inX[pair.left ];
    inlined.right = inX[pair.right];

    CUDA_ASSERT( inlined.left || inlined.right );

    inlinedPairs[gid] = inlined;
}

//-----------------------------------------------------------
template<bool UseLP>
__global__ void CudaCompressTable( const uint32* entryCount, const uint32* inLEntries, const Pair* matches, uint32* outREntries, const uint32 bitShift )
{
    const uint32 gid = blockIdx.x * blockDim.x + threadIdx.x;

    if( gid >= *entryCount )
        return;

    const Pair pair = matches[gid];

    const uint32 x0 = inLEntries[pair.left ];
    const uint32 x1 = inLEntries[pair.right];

    // Convert to linepoint   
    if constexpr ( UseLP )         
        outREntries[gid] = (uint32)CudaSquareToLinePoint64( x1 >> bitShift, x0 >> bitShift );
    else
        outREntries[gid] =  ((x1 >> bitShift) << (32-bitShift) ) | (x0 >> bitShift);
}

//-----------------------------------------------------------
void InlineTable( CudaK32PlotContext& cx, const uint32* devInX, hipStream_t stream )
{
    static_assert( alignof( Pair ) == sizeof( uint32 ) );

    const bool isCompressedInput = cx.gCfg->compressionLevel > 0 && (uint32)cx.table <= cx.gCfg->numDroppedTables;

    const uint32 kthreads = 256;
    const uint32 kblocks  = CDiv( BBCU_BUCKET_ALLOC_ENTRY_COUNT, (int)kthreads );
    
    if( isCompressedInput )
    {
        const bool   isFinalTable = cx.table == TableId::Table1 + (TableId)cx.gCfg->numDroppedTables;
        const uint32 bitShift     = ( isFinalTable && cx.gCfg->numDroppedTables > 1 ) ? 0 : BBCU_K - cx.gCfg->compressedEntryBits;

        if( isFinalTable )
            CudaCompressTable<true><<<kblocks, kthreads, 0, stream>>>( cx.devMatchCount, devInX, cx.devMatches, cx.devCompressedXs, bitShift );
        else
            CudaCompressTable<false><<<kblocks, kthreads, 0, stream>>>( cx.devMatchCount, devInX, cx.devMatches, cx.devCompressedXs, bitShift );
    }
    else
    {
        CudaInlineTable<<<kblocks, kthreads, 0, stream>>>( cx.devMatchCount, devInX, cx.devMatches, cx.devInlinedXs );
    }
}

//-----------------------------------------------------------
void CudaK32PlotDownloadBucket( CudaK32PlotContext& cx )
{
    const bool   writeVertical  = CudaK32PlotIsOutputInterleaved( cx );
    const size_t metaMultiplier = GetTableMetaMultiplier( cx.table );

    const bool   downloadCompressed   = cx.table > TableId::Table1 && (uint32)cx.table <= cx.gCfg->numDroppedTables;
    const bool   downloadInlinedPairs = !downloadCompressed && (uint32)cx.table == cx.gCfg->numDroppedTables+1;

    uint32* hostY        = cx.hostY;
    uint32* hostMeta     = cx.hostMeta;

    uint32* hostPairsL   = cx.hostTableL; //cx.hostBackPointers[6].left;
    uint16* hostPairsR   = cx.hostTableR; //cx.hostBackPointers[6].right;
    Pair*   t2HostPairs  = (Pair*)cx.hostBackPointers[4].left;

    const size_t startOffset  = cx.bucket * ( writeVertical ? BBCU_MAX_SLICE_ENTRY_COUNT : BBCU_BUCKET_ALLOC_ENTRY_COUNT );  // vertical: offset to starting col. horizontal: to starting row
    const size_t width        = BBCU_MAX_SLICE_ENTRY_COUNT;
    const size_t height       = BBCU_BUCKET_COUNT;
    const size_t dstStride    = writeVertical ? BBCU_BUCKET_ALLOC_ENTRY_COUNT : BBCU_MAX_SLICE_ENTRY_COUNT;
    const size_t srcStride    = BBCU_MAX_SLICE_ENTRY_COUNT;

    cx.yOut.Download2DT<uint32>( hostY + startOffset, width, height, dstStride, srcStride, cx.computeStream );

    // Metadata
    if( metaMultiplier > 0 )
    {
        const size_t metaSizeMultiplier = metaMultiplier == 3 ? 4 : metaMultiplier;
        const size_t metaSize           = sizeof( uint32 ) * metaSizeMultiplier;
        
        const size_t  metaSrcStride = srcStride * metaSize;
        const size_t  metaDstStride = dstStride * sizeof( K32Meta4 );
        const size_t  metaWidth     = width * metaSize;
              uint32* meta          = hostMeta + startOffset * 4;

        cx.metaOut.Download2D( meta, metaWidth, height, metaDstStride, metaSrcStride, cx.computeStream );
    }

    if( cx.table > TableId::Table1 )
    {
        if( downloadInlinedPairs )
        {
            cx.xPairsOut.Download2DT<Pair>( t2HostPairs + startOffset, width, height, dstStride, srcStride, cx.computeStream );
        }
        else
        {
            cx.pairsLOut.Download2DT<uint32>( hostPairsL + startOffset, width, height, dstStride, srcStride, cx.computeStream );

            if( !downloadCompressed )
                cx.pairsROut.Download2DT<uint16>( hostPairsR + startOffset, width, height, dstStride, srcStride, cx.computeStream );
        }
    }
}

//-----------------------------------------------------------
void UploadBucketForTable( CudaK32PlotContext& cx, const uint64 bucket )
{
    const TableId rTable  = cx.table;
    const TableId inTable = rTable - 1;

    uint32 metaMultiplier = GetTableMetaMultiplier( inTable );

    const uint32  inIdx        = CudaK32PlotGetInputIndex( cx );
    const bool    readVertical = CudaK32PlotIsOutputInterleaved( cx );

    const uint32* hostY        = cx.hostY;
    const uint32* hostMeta     = cx.hostMeta;
    const uint32* hostPairsL   = cx.hostTableL; //cx.hostBackPointers[6].left;
    const uint16* hostPairsR   = cx.hostTableR; //cx.hostBackPointers[6].right;

    const bool   uploadCompressed   = cx.table > TableId::Table2 && (uint32)cx.table-1 <= cx.gCfg->numDroppedTables;
    const bool   uploadInlinedPairs = !uploadCompressed && (uint32)cx.table == cx.gCfg->numDroppedTables+2;
    const Pair*  t2HostPairs        = (Pair*)cx.hostBackPointers[4].left; // Table 2 will use table 5, and overflow onto 6

    uint32 stride = BBCU_BUCKET_ALLOC_ENTRY_COUNT;          // Start as vertical
    size_t offset = (size_t)bucket * BBCU_MAX_SLICE_ENTRY_COUNT;

    if( !readVertical )
    {
        // Adjust to starting row
        stride = BBCU_MAX_SLICE_ENTRY_COUNT;
        offset = (size_t)bucket * BBCU_BUCKET_ALLOC_ENTRY_COUNT;
    }

    hipStream_t mainStream  = cx.computeStream;
    hipStream_t metaStream  = cx.computeStream;//B;
    hipStream_t pairsStream = cx.computeStream;//C;

    const uint32* counts = &cx.bucketSlices[inIdx][0][bucket];

    cx.yIn.UploadArrayT<uint32>( hostY + offset, BBCU_BUCKET_COUNT, stride, BBCU_BUCKET_COUNT, counts, cx.computeStream );

    // Upload pairs, also
    if( cx.table > TableId::Table2 )
    {
        if( uploadInlinedPairs )
        {
            cx.xPairsIn.UploadArrayT<Pair>( t2HostPairs + offset, BBCU_BUCKET_COUNT, stride, BBCU_BUCKET_COUNT, counts, pairsStream );
        }
        else
        {
            cx.pairsLIn.UploadArrayT<uint32>( hostPairsL + offset, BBCU_BUCKET_COUNT, stride, BBCU_BUCKET_COUNT, counts, pairsStream );

            if( !uploadCompressed )
                cx.pairsRIn.UploadArrayT<uint16>( hostPairsR + offset, BBCU_BUCKET_COUNT, stride, BBCU_BUCKET_COUNT, counts, pairsStream );
        }
    }
    
    // Meta
    if( metaMultiplier > 0 )
    {
        const size_t metaSizeMultiplier = metaMultiplier == 3 ? 4 : metaMultiplier;
        const size_t metaSize           = sizeof( uint32 ) * metaSizeMultiplier;

        auto actualMetaStream = inTable == TableId::Table1 ? cx.computeStream : metaStream;
        cx.metaIn.UploadArray( hostMeta + offset * 4, BBCU_BUCKET_COUNT, metaSize, stride * sizeof( K32Meta4 ), BBCU_BUCKET_COUNT, counts, actualMetaStream );
    }
}


///
/// Allocations
///
//-----------------------------------------------------------
void AllocBuffers( CudaK32PlotContext& cx )
{
    // Determine initially the largest required size

    const size_t alignment = bbclamp<size_t>( SysHost::GetPageSize(), sizeof( K32Meta4 ), 4096 );
    cx.allocAlignment     = alignment;
    cx.pinnedAllocSize    = 0;
    cx.hostTableAllocSize = 0;
    cx.hostTempAllocSize  = 0;
    cx.devAllocSize       = 0;

    // Gather the size needed first
    {
        CudaK32AllocContext acx = {};

        acx.alignment = alignment;
        acx.dryRun    = true;
        
        DummyAllocator pinnedAllocator;
        DummyAllocator hostTableAllocator;
        DummyAllocator hostTempAllocator;
        DummyAllocator devAllocator;

        acx.pinnedAllocator    = &pinnedAllocator;
        acx.hostTableAllocator = &hostTableAllocator;
        acx.hostTempAllocator  = &hostTempAllocator;
        acx.devAllocator       = &devAllocator;

        AllocateP1Buffers( cx, acx );

        cx.pinnedAllocSize    = pinnedAllocator   .Size();
        cx.hostTableAllocSize = hostTableAllocator.Size();
        cx.hostTempAllocSize  = hostTempAllocator .Size();
        cx.devAllocSize       = devAllocator      .Size();

        /// Phase 2
        pinnedAllocator    = {};
        hostTableAllocator = {};
        hostTempAllocator  = {};
        devAllocator       = {};

        CudaK32PlotPhase2AllocateBuffers( cx, acx );

        cx.pinnedAllocSize    = std::max( cx.pinnedAllocSize   , pinnedAllocator   .Size() );
        cx.hostTableAllocSize = std::max( cx.hostTableAllocSize, hostTableAllocator.Size() );
        cx.hostTempAllocSize  = std::max( cx.hostTempAllocSize , hostTempAllocator .Size() );
        cx.devAllocSize       = std::max( cx.devAllocSize      , devAllocator      .Size() );

        /// Phase 3
        pinnedAllocator    = {};
        hostTableAllocator = {};
        hostTempAllocator  = {};
        devAllocator       = {};

        CudaK32PlotPhase3AllocateBuffers( cx, acx );

        cx.pinnedAllocSize    = std::max( cx.pinnedAllocSize   , pinnedAllocator   .Size() );
        cx.hostTableAllocSize = std::max( cx.hostTableAllocSize, hostTableAllocator.Size() );
        cx.hostTempAllocSize  = std::max( cx.hostTempAllocSize , hostTempAllocator .Size() );
        cx.devAllocSize       = std::max( cx.devAllocSize      , devAllocator      .Size() );
    }

    size_t totalPinnedSize = cx.pinnedAllocSize + cx.hostTempAllocSize;
    size_t totalHostSize   = cx.hostTableAllocSize + totalPinnedSize;
    Log::Line( "Kernel RAM required       : %-12llu bytes ( %-9.2lf MiB or %-6.2lf GiB )", totalPinnedSize,
                   (double)totalPinnedSize BtoMB, (double)totalPinnedSize BtoGB );

    Log::Line( "Intermediate RAM required : %-12llu bytes ( %-9.2lf MiB or %-6.2lf GiB )", cx.pinnedAllocSize,
                   (double)cx.pinnedAllocSize BtoMB, (double)cx.pinnedAllocSize BtoGB );

    Log::Line( "Host RAM required         : %-12llu bytes ( %-9.2lf MiB or %-6.2lf GiB )", cx.hostTableAllocSize,
                    (double)cx.hostTableAllocSize BtoMB, (double)cx.hostTableAllocSize BtoGB );

    Log::Line( "Total Host RAM required   : %-12llu bytes ( %-9.2lf MiB or %-6.2lf GiB )", totalHostSize,
                    (double)totalHostSize BtoMB, (double)totalHostSize BtoGB );

    Log::Line( "GPU RAM required          : %-12llu bytes ( %-9.2lf MiB or %-6.2lf GiB )", cx.devAllocSize,
                   (double)cx.devAllocSize BtoMB, (double)cx.devAllocSize BtoGB );

    Log::Line( "Allocating buffers" );
    // Now actually allocate the buffers
    CudaErrCheck( hipHostMalloc( &cx.pinnedBuffer, cx.pinnedAllocSize, hipHostMallocDefault ) );

    #if _DEBUG
        cx.hostBufferTables = bbvirtallocboundednuma<byte>( cx.hostTableAllocSize );
    #else
        #if !_WIN32
        // if( cx.downloadDirect )
            CudaErrCheck( hipHostMalloc( &cx.hostBufferTables, cx.hostTableAllocSize, hipHostMallocDefault ) );
        // else
        // {
        //     // #TODO: On windows, first check if we have enough shared memory (512G)? 
        //     //        and attempt to alloc that way first. Otherwise, use intermediate pinned buffers.
        #else
            cx.hostBufferTables = bbvirtallocboundednuma<byte>( cx.hostTableAllocSize );
        #endif
        // }
    #endif

    //CudaErrCheck( hipHostMalloc( &cx.hostBufferTables, cx.hostTableAllocSize, hipHostMallocDefault ) );

    cx.hostBufferTemp = nullptr;
#if _DEBUG
    cx.hostBufferTemp   = bbvirtallocboundednuma<byte>( cx.hostTempAllocSize );
#endif
    if( cx.hostBufferTemp == nullptr )
        CudaErrCheck( hipHostMalloc( &cx.hostBufferTemp, cx.hostTempAllocSize, hipHostMallocDefault ) );

    CudaErrCheck( hipMalloc( &cx.deviceBuffer, cx.devAllocSize ) );

    // Warm start
    if( true )
    {
        FaultMemoryPages::RunJob( *cx.threadPool, cx.threadPool->ThreadCount(), cx.pinnedBuffer, cx.pinnedAllocSize );
        FaultMemoryPages::RunJob( *cx.threadPool, cx.threadPool->ThreadCount(), cx.hostBufferTables, cx.hostTableAllocSize );
        FaultMemoryPages::RunJob( *cx.threadPool, cx.threadPool->ThreadCount(), cx.hostBufferTemp, cx.hostTempAllocSize );
    }

    {
        CudaK32AllocContext acx = {};

        acx.alignment = alignment;
        acx.dryRun    = false;
        
        StackAllocator pinnedAllocator   ( cx.pinnedBuffer    , cx.pinnedAllocSize    );
        StackAllocator hostTableAllocator( cx.hostBufferTables, cx.hostTableAllocSize );
        StackAllocator hostTempAllocator ( cx.hostBufferTemp  , cx.hostTempAllocSize  );
        StackAllocator devAllocator      ( cx.deviceBuffer    , cx.devAllocSize       );

        acx.pinnedAllocator    = &pinnedAllocator;
        acx.hostTableAllocator = &hostTableAllocator;
        acx.hostTempAllocator  = &hostTempAllocator;
        acx.devAllocator       = &devAllocator;
        AllocateP1Buffers( cx, acx );

        pinnedAllocator   .PopToMarker( 0 );
        hostTableAllocator.PopToMarker( 0 );
        hostTempAllocator .PopToMarker( 0 );
        devAllocator      .PopToMarker( 0 );
        CudaK32PlotPhase2AllocateBuffers( cx, acx );

        pinnedAllocator   .PopToMarker( 0 );
        hostTableAllocator.PopToMarker( 0 );
        hostTempAllocator .PopToMarker( 0 );
        devAllocator      .PopToMarker( 0 );
        CudaK32PlotPhase3AllocateBuffers( cx, acx );
    }
}

//-----------------------------------------------------------
void AllocateP1Buffers( CudaK32PlotContext& cx, CudaK32AllocContext& acx )
{
    const size_t alignment = acx.alignment;

    const bool isCompressed = cx.gCfg->compressionLevel > 0;

    // #TODO: Re-optimize usage here again for windows running 256G
    /// Host allocations
    {
        // Temp allocations are pinned host buffers that can be re-used for other means in different phases.
        // This is roughly equivalent to temp2 dir during disk plotting.
        cx.hostY    = acx.hostTempAllocator->CAlloc<uint32>( BBCU_TABLE_ALLOC_ENTRY_COUNT, alignment );
        cx.hostMeta = acx.hostTempAllocator->CAlloc<uint32>( BBCU_TABLE_ALLOC_ENTRY_COUNT * BBCU_HOST_META_MULTIPLIER, alignment );

        const size_t markingTableBitFieldSize = GetMarkingTableBitFieldSize();

        cx.hostMarkingTables[0] = nullptr;
        cx.hostMarkingTables[1] = isCompressed ? nullptr : acx.hostTableAllocator->AllocT<uint64>( markingTableBitFieldSize, alignment );
        cx.hostMarkingTables[2] = acx.hostTableAllocator->AllocT<uint64>( markingTableBitFieldSize, alignment );
        cx.hostMarkingTables[3] = acx.hostTableAllocator->AllocT<uint64>( markingTableBitFieldSize, alignment );
        cx.hostMarkingTables[4] = acx.hostTableAllocator->AllocT<uint64>( markingTableBitFieldSize, alignment );
        cx.hostMarkingTables[5] = acx.hostTableAllocator->AllocT<uint64>( markingTableBitFieldSize, alignment );

    
        // NOTE: The first table has their values inlines into the backpointers of the next table
        cx.hostBackPointers[0] = {};

        const TableId firstTable = TableId::Table2 + (TableId)cx.gCfg->numDroppedTables;
        
        Pair* firstTablePairs = acx.hostTableAllocator->CAlloc<Pair>( BBCU_TABLE_ALLOC_ENTRY_COUNT, alignment );
        cx.hostBackPointers[(int)firstTable] = { (uint32*)firstTablePairs, nullptr };

        for( TableId table = firstTable + 1; table <= TableId::Table7; table++ )
            cx.hostBackPointers[(int)table] = { acx.hostTableAllocator->CAlloc<uint32>( BBCU_TABLE_ALLOC_ENTRY_COUNT, alignment ), acx.hostTableAllocator->CAlloc<uint16>( BBCU_TABLE_ALLOC_ENTRY_COUNT, alignment ) };

        cx.hostTableL       = cx.hostBackPointers[6].left;     // Also used for Table 7
        cx.hostTableR       = cx.hostBackPointers[6].right;
        cx.hostTableSortedL = cx.hostBackPointers[5].left;
        cx.hostTableSortedR = cx.hostBackPointers[5].right;
    }

    /// Device & Pinned allocations
    {
        // #NOTE: The R pair is allocated as uint32 because for table 2 we want to download them as inlined x's, so we need 2 uint32 buffers
        /// Device/Pinned allocations
        // cx.yOut    = cx.gpuDownloadStream[0]->CreateDownloadBufferT<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
        // cx.metaOut = cx.gpuDownloadStream[0]->CreateDownloadBufferT<K32Meta4>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
        cx.yOut    = cx.gpuDownloadStream[0]->CreateDirectDownloadBuffer<uint32>  ( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, alignment, acx.dryRun );
        cx.metaOut = cx.gpuDownloadStream[0]->CreateDirectDownloadBuffer<K32Meta4>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, alignment, acx.dryRun );

        // These download buffers share the same backing buffers
        {
            const size_t devMarker    = acx.devAllocator->Size();
            const size_t pinnedMarker = acx.pinnedAllocator->Size();

            cx.pairsLOut = cx.gpuDownloadStream[0]->CreateDirectDownloadBuffer<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, alignment, acx.dryRun );
            cx.pairsROut = cx.gpuDownloadStream[0]->CreateDirectDownloadBuffer<uint16>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, alignment, acx.dryRun );

            acx.devAllocator->PopToMarker( devMarker );
            acx.pinnedAllocator->PopToMarker( pinnedMarker );

            // Allocate Pair at the end, to ensure we grab the highest value
            cx.xPairsOut = cx.gpuDownloadStream[0]->CreateDownloadBufferT<Pair>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
        }

        // These download buffers share the same backing buffers
        {
            const size_t devMarker    = acx.devAllocator->Size();
            const size_t pinnedMarker = acx.pinnedAllocator->Size();

            cx.sortedPairsLOut = cx.gpuDownloadStream[0]->CreateDownloadBufferT<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
            cx.sortedPairsROut = cx.gpuDownloadStream[0]->CreateDownloadBufferT<uint16>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );

            acx.devAllocator->PopToMarker( devMarker );
            acx.pinnedAllocator->PopToMarker( pinnedMarker );

            // Allocate Pair at the end, to ensure we grab the highest value
            cx.sortedXPairsOut = cx.gpuDownloadStream[0]->CreateDownloadBufferT<Pair>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
        }

        cx.yIn    = cx.gpuUploadStream[0]->CreateUploadBufferT<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
        cx.metaIn = cx.gpuUploadStream[0]->CreateUploadBufferT<K32Meta4>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );

        // These uploaded buffers share the same backing buffers
        {
            const size_t devMarker    = acx.devAllocator->Size();
            const size_t pinnedMarker = acx.pinnedAllocator->Size();

            cx.pairsLIn = cx.gpuUploadStream[0]->CreateUploadBufferT<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
            cx.pairsRIn = cx.gpuUploadStream[0]->CreateUploadBufferT<uint16>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );

            acx.devAllocator->PopToMarker( devMarker );
            acx.pinnedAllocator->PopToMarker( pinnedMarker );

            // Allocate Pair at the end, to ensure we grab the highest value
            cx.xPairsIn = cx.gpuUploadStream[0]->CreateUploadBufferT<Pair>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, *acx.devAllocator, *acx.pinnedAllocator, alignment, acx.dryRun );
        }

        /// Device-only allocations
        if( acx.dryRun )
        {
            cx.devSortTmpAllocSize = 0;
            hipcub::DeviceRadixSort::SortPairs<uint32, uint32>( nullptr, cx.devSortTmpAllocSize, nullptr, nullptr, nullptr, nullptr, BBCU_BUCKET_ALLOC_ENTRY_COUNT );
        }

        cx.devSortTmp         = acx.devAllocator->AllocT<byte>( cx.devSortTmpAllocSize, alignment );

        cx.devYWork           = acx.devAllocator->CAlloc<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, alignment );
        cx.devMetaWork        = acx.devAllocator->CAlloc<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT * BBCU_HOST_META_MULTIPLIER, alignment );
        cx.devXInlineInput    = acx.devAllocator->CAlloc<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, alignment );   // #TODO: Maybe we can avoid this allocation?
        cx.devMatches         = acx.devAllocator->CAlloc<Pair>  ( BBCU_BUCKET_ALLOC_ENTRY_COUNT, alignment );
        cx.devInlinedXs       = acx.devAllocator->CAlloc<Pair>  ( BBCU_BUCKET_ALLOC_ENTRY_COUNT, alignment );

        cx.devSortKey         = acx.devAllocator->CAlloc<uint32>( BBCU_BUCKET_ALLOC_ENTRY_COUNT, alignment );
        cx.devChaChaInput     = (uint32*)acx.devAllocator->AllocT<byte>( 64, alignment );
        cx.devGroupBoundaries = acx.devAllocator->CAlloc<uint32>( CU_MAX_BC_GROUP_BOUNDARIES, alignment );
        cx.devMatchCount      = acx.devAllocator->CAlloc<uint32>( 1 );
        cx.devGroupCount      = acx.devAllocator->CAlloc<uint32>( 1 );
        cx.devBucketCounts    = acx.devAllocator->CAlloc<uint32>( BBCU_BUCKET_COUNT, alignment );
        cx.devSliceCounts     = acx.devAllocator->CAlloc<uint32>( BBCU_BUCKET_COUNT * BBCU_BUCKET_COUNT, alignment );


        /// Pinned-only allocations
        cx.hostMatchCount   = acx.pinnedAllocator->CAlloc<uint32>( 1, alignment );
        cx.hostBucketCounts = acx.pinnedAllocator->CAlloc<uint32>( BBCU_BUCKET_COUNT, alignment );
        cx.hostBucketSlices = acx.pinnedAllocator->CAlloc<uint32>( BBCU_BUCKET_COUNT * BBCU_BUCKET_COUNT, alignment );
    }
}


///
/// Debug
///
#if _DEBUG

void DbgWritePairs( CudaK32PlotContext& cx, const TableId table )
{
    const TableId earliestTable = TableId::Table1 + (TableId)cx.gCfg->numDroppedTables+1;
    if( table < earliestTable )
        return;

    char lPath[512];
    char rPath[512];

    Log::Line( "[DEBUG] Writing pairs to disk..." );
    {
        sprintf( lPath, "%st%d.l.tmp", DBG_BBCU_DBG_DIR, (int)table+1 );
        sprintf( rPath, "%st%d.r.tmp", DBG_BBCU_DBG_DIR, (int)table+1 );

        const uint64 entryCount = cx.tableEntryCounts[(int)table];
        const Pairs  pairs      = cx.hostBackPointers[(int)table];

        int err;

        if( table == earliestTable )
        {
            FatalIf( !IOJob::WriteToFile( lPath, pairs.left, sizeof( Pair ) * entryCount, err ),
                "Failed to write table pairs: %d", err );
        }
        else
        {
            FatalIf( !IOJob::WriteToFile( lPath, pairs.left, sizeof( uint32 ) * entryCount, err ),
                "Failed to write table L pairs: %d", err );

            // if( (uint32)table > cx.gCfg->numDroppedTables )
                FatalIf( !IOJob::WriteToFile( rPath, pairs.right, sizeof( uint16 ) * entryCount, err),
                    "Failed to write table R pairs: %d", err );
        }
    }

    // if( cx.table == TableId::Table7 )
    // {
    //     // Now write our context data
    //     Log::Line( "[DEBUG] Writing context file." );
    //     FileStream contxetFile;
    //     sprintf( lPath, "%scontext.tmp", DBG_BBCU_DBG_DIR );
    //     FatalIf( !contxetFile.Open( lPath, FileMode::Create, FileAccess::Write ), "Failed to open context file." );
    //     FatalIf( contxetFile.Write( &cx, sizeof( CudaK32PlotContext ) ) != (ssize_t)sizeof( CudaK32PlotContext ), "Failed to write context data." );
    //     contxetFile.Close();
    // }
    Log::Line( "[DEBUG] Done." );
}

void DbgWriteContext( CudaK32PlotContext& cx )
{
    char path[512];
    
    // Now write our context data
    Log::Line( "[DEBUG] Writing context file." );
    FileStream contxetFile;
    sprintf( path, "%scontext.tmp", DBG_BBCU_DBG_DIR );
    FatalIf( !contxetFile.Open( path, FileMode::Create, FileAccess::Write ), "Failed to open context file." );
    FatalIf( contxetFile.Write( &cx, sizeof( CudaK32PlotContext ) ) != (ssize_t)sizeof( CudaK32PlotContext ), "Failed to write context data." );
    
    contxetFile.Close();
    
    Log::Line( "[DEBUG] Done." );
}

void DbgLoadContextAndPairs( CudaK32PlotContext& cx, bool loadTables )
{
    char lPath[512];
    char rPath[512];

    // Log::Line( "[DEBUG] Loading table pairs..." );
    {
        Log::Line( "[DEBUG] Reading context" );
        CudaK32PlotContext tmpCx = {};

        FileStream contxetFile;
        sprintf( lPath, "%scontext.tmp", DBG_BBCU_DBG_DIR );
        FatalIf( !contxetFile.Open( lPath, FileMode::Open, FileAccess::Read ), "Failed to open context file." );
        FatalIf( contxetFile.Read( &tmpCx, sizeof( CudaK32PlotContext ) ) != (ssize_t)sizeof( CudaK32PlotContext ), "Failed to read context data." );
        contxetFile.Close();

        memcpy( cx.bucketCounts, tmpCx.bucketCounts, sizeof( tmpCx.bucketCounts ) );
        memcpy( cx.bucketSlices, tmpCx.bucketSlices, sizeof( tmpCx.bucketSlices ) );
        memcpy( cx.tableEntryCounts, tmpCx.tableEntryCounts, sizeof( tmpCx.tableEntryCounts ) );        
    }
    
    if( !loadTables )
        return;

    for( TableId table = TableId::Table2; table <= TableId::Table7; table++ )
    {
        Log::Line( "[DEBUG] Loading table %d", (int)table+1 );

        sprintf( lPath, "%st%d.l.tmp", DBG_BBCU_DBG_DIR, (int)table+1 );
        sprintf( rPath, "%st%d.r.tmp", DBG_BBCU_DBG_DIR, (int)table+1 );

        const uint64  entryCount = cx.tableEntryCounts[(int)table];
              Pairs&  pairs      = cx.hostBackPointers[(int)table];


        int err;
        pairs.left = (uint32*)IOJob::ReadAllBytesDirect( lPath, err );
        FatalIf( pairs.left == nullptr, "Failed to read table L pairs: %d", err );

        pairs.right = (uint16*)IOJob::ReadAllBytesDirect( rPath, err );
        FatalIf( pairs.right == nullptr, "Failed to read table R pairs: %d", err );
    }
}

void DbgLoadTablePairs( CudaK32PlotContext& cx, const TableId table, bool copyToPinnedBuffer )
{
    char lPath[512];
    char rPath[512];

    const TableId earliestTable = TableId::Table1 + (TableId)cx.gCfg->numDroppedTables+1;
    if( table < earliestTable )
        return;

    // for( TableId table = TableId::Table2; table <= TableId::Table7; table++ )
    {
        Log::Line( "[DEBUG] Loading table %d", (int)table + 1 );

        sprintf( lPath, "%st%d.l.tmp", DBG_BBCU_DBG_DIR, (int)table + 1 );
        sprintf( rPath, "%st%d.r.tmp", DBG_BBCU_DBG_DIR, (int)table + 1 );

        const uint64 entryCount = cx.tableEntryCounts[(int)table];
        // cx.hostBackPointers[(int)table].left  = bbcvirtallocbounded<uint32>( entryCount );
        // cx.hostBackPointers[(int)table].right = bbcvirtallocbounded<uint16>( entryCount );
        Pairs& pairs = cx.hostBackPointers[(int)table];

        int err;

        if( table == earliestTable )
        {
            FatalIf( !IOJob::ReadFromFile( lPath, pairs.left, entryCount * sizeof( Pair ), err ), "Failed to read table X pairs: %d", err );
        }
        else
        {
            FatalIf( !IOJob::ReadFromFile( lPath, pairs.left , entryCount * sizeof( uint32 ), err ), "Failed to read table L pairs: %d", err );
            
            // if( (uint32)table > cx.gCfg->numDroppedTables )
                FatalIf( !IOJob::ReadFromFile( rPath, pairs.right, entryCount * sizeof( uint16 ), err ), "Failed to read table R pairs: %d", err );
        }

        // We expect table 7 to also be found in these buffers, so copy it
        // if( table == TableId::Table7 )
        if( copyToPinnedBuffer )
        {
            bbmemcpy_t( cx.hostTableSortedL, pairs.left , entryCount );
            bbmemcpy_t( cx.hostTableSortedR, pairs.right, entryCount );
        }
    }

    Log::Line( "[DEBUG] Done." );
}


void DbgLoadMarks( CudaK32PlotContext& cx )
{
    char path[512];

    // const size_t tableSize = ((1ull << BBCU_K) / 64) * sizeof(uint64);
    Log::Line( "[DEBUG] Loadinging marking tables" );

    const TableId startTable = TableId::Table2 + cx.gCfg->numDroppedTables; 

    for( TableId table = startTable; table < TableId::Table7; table++ )
    {
        sprintf( path, "%smarks%d.tmp", DBG_BBCU_DBG_DIR, (int)table+1 );

        int err = 0;
        cx.hostMarkingTables[(int)table] = (uint64*)IOJob::ReadAllBytesDirect( path, err );
    }

    Log::Line( "[DEBUG] Done." );
}

void DbgPruneTable( CudaK32PlotContext& cx, const TableId rTable )
{
    const size_t MarkingTableSize = 1ull << 32;
    byte* bytefield = bbvirtalloc<byte>( MarkingTableSize );
    memset( bytefield, 0, MarkingTableSize );
    
    std::atomic<uint64> totalPrunedEntryCount = 0;

    ThreadPool& pool = DbgGetThreadPool( cx );
    AnonMTJob::Run( pool, [&]( AnonMTJob* self ){

        const uint64 rEntryCount = cx.tableEntryCounts[(int)rTable];
        {
            uint64 count, offset, end;
            GetThreadOffsets( self, rEntryCount, count, offset, end );

            auto marks = bytefield;
            Pairs rTablePairs = cx.hostBackPointers[(int)rTable];

            for( uint64 i = offset; i < end; i++ )
            {
                const uint32 l = rTablePairs.left[i];
                const uint32 r = l + rTablePairs.right[i];
                
                marks[l] = 1;
                marks[r] = 1;
            }

            self->SyncThreads();

                  uint64 localPrunedEntryCount = 0;
            const uint64 lEntryCount           = cx.tableEntryCounts[(int)rTable-1];

            GetThreadOffsets( self, lEntryCount, count, offset, end );
            for( uint64 i = offset; i < end; i++ )
            {
                if( marks[i] == 1 )
                    localPrunedEntryCount++;
            }

            totalPrunedEntryCount += localPrunedEntryCount;
        }
    });

    const uint64 prunedEntryCount = totalPrunedEntryCount.load();
    const uint64 lEntryCount      = cx.tableEntryCounts[(int)rTable-1];
    Log::Line( "Table %u pruned entry count: %llu / %llu ( %.2lf %% )", rTable, 
        prunedEntryCount, lEntryCount, prunedEntryCount / (double)lEntryCount * 100.0 );

    bbvirtfree( bytefield );
}

void DbgPruneTableBuckets( CudaK32PlotContext& cx, const TableId rTable )
{
    const size_t MarkingTableSize = 1ull << 32;
    byte* bytefield = bbvirtalloc<byte>( MarkingTableSize );
    memset( bytefield, 0, MarkingTableSize );
    
    std::atomic<uint64> totalPrunedEntryCount = 0;

    AnonMTJob::Run( *_dbgThreadPool, [&]( AnonMTJob* self ){

        auto  marks = bytefield;

        Pairs rTablePairs = cx.hostBackPointers[6];

        for( uint32 bucket = 0; bucket < BBCU_BUCKET_COUNT; bucket++ )
        {
            const uint64 rEntryCount = cx.bucketCounts[(int)rTable][bucket];

            uint64 count, offset, end;
            GetThreadOffsets( self, rEntryCount, count, offset, end );

            for( uint64 i = offset; i < end; i++ )
            {
                const uint32 l = rTablePairs.left[i];
                const uint32 r = l + rTablePairs.right[i];
                
                marks[l] = 1;
                marks[r] = 1;
            }

            rTablePairs.left  += BBCU_BUCKET_ALLOC_ENTRY_COUNT;
            rTablePairs.right += BBCU_BUCKET_ALLOC_ENTRY_COUNT;
        }

        self->SyncThreads();

        {
                  uint64 localPrunedEntryCount = 0;
            const uint64 lEntryCount           = cx.tableEntryCounts[(int)rTable-1];

            uint64 count, offset, end;
            GetThreadOffsets( self, lEntryCount, count, offset, end );

            for( uint64 i = offset; i < end; i++ )
            {
                if( marks[i] == 1 )
                    localPrunedEntryCount++;
            }

            totalPrunedEntryCount += localPrunedEntryCount;
        }
    });

    const uint64 prunedEntryCount = totalPrunedEntryCount.load();
    const uint64 lEntryCount      = cx.tableEntryCounts[(int)rTable-1];
    Log::Line( "Table %u pruned entry count: %llu / %llu ( %.2lf %% )", rTable, 
        prunedEntryCount, lEntryCount, prunedEntryCount / (double)lEntryCount * 100.0 );

    bbvirtfree( bytefield );
}

#endif // _DEBUG


